#include "hip/hip_runtime.h"
/*
  Assignment 2 - Image Blurring
  Author: Luis Carlos Arias Camacho
  Student ID: A01364808
 */

#include <iostream>
#include <cstdio>
#include <cmath>
#include <chrono>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "common.h"
#include <hip/hip_runtime.h>

#define default_input_image "image.jpg"
#define size1 5
#define size2 11

using namespace std;

__global__ void blur_kernel(unsigned char* input_Image, unsigned char* output_Image, int width, int height, int colorWidthStep, int bM_size){

  //pixel margin for blur matrix
  const int margin = floor(bM_size / 2.0);
  const float multConstant = bM_size * bM_size;

	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	// Only pixels out of margin
	if ((xIndex < width) && (yIndex < height)) {

		int output_index = yIndex * colorWidthStep + (3 * xIndex);

		//Output pixels
		float blue = 0;
		float green = 0;
		float red = 0;

		//Pixels inside blur section
		if ((xIndex >= margin) && (xIndex < width - margin) && (yIndex >= margin) && (yIndex < height - margin)) {

			int index = 0;

			//Calculate blur average
			for (int i = xIndex - margin; i < xIndex + margin + 1; i++) {
				for (int j = yIndex - margin; j < yIndex + margin + 1; j++) {
					index = j * colorWidthStep + (3 * i);
					blue += input_Image[index];
					green += input_Image[index + 1];
					red += input_Image[index + 2];
				}
			}
			blue = blue / multConstant;
			green = green / multConstant;
			red = red / multConstant;
		} else {
			//If pixels are in margin range
			int input_index = yIndex * colorWidthStep + (3 * xIndex);
			blue = input_Image[input_index];
			green = input_Image[input_index + 1];
			red = input_Image[input_index + 2];
		}
		output_Image[output_index] = static_cast<unsigned char>(blue);
		output_Image[output_index + 1] = static_cast<unsigned char>(green);
		output_Image[output_index + 2] = static_cast<unsigned char>(red);
	}

}

void blur_GPU(const cv::Mat& input, cv::Mat& output, int blurMatrix_size){

	cout << "Input image step: " << input.step << " rows: " << input.rows << " cols: " << input.cols << endl;

	// Calculate total number of bytes of input and output image
	// Step = cols * number of colors
	size_t inputBytes = input.step * input.rows;
	size_t outputBytes = output.step * output.rows;

	unsigned char *d_input, *d_output;

	// Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input, inputBytes), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output, outputBytes), "CUDA Malloc Failed");

	// Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input, input.ptr(), inputBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

	// Specify a reasonable block size
	const dim3 block(16, 16);

	// Calculate grid size to cover the whole image
	const dim3 grid((int)ceil((float)input.cols / block.x), (int)ceil((float)input.rows/ block.y));
	printf("blur_kernel<<<(%d, %d) , (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);

	chrono::duration<float, std::milli> duration_ms = chrono::high_resolution_clock::duration::zero();
	auto start_gpu =  chrono::high_resolution_clock::now();
	// Launch the color conversion kernel
	blur_kernel <<<grid, block >>>(d_input, d_output, input.cols, input.rows, static_cast<int>(input.step), blurMatrix_size);

	auto end_gpu =  chrono::high_resolution_clock::now();
	duration_ms = end_gpu - start_gpu;
	printf("Image blur elapsed %f ms in GPU with a blur matrix of %dx%d\n", duration_ms.count(), blurMatrix_size, blurMatrix_size);

	// Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

	// Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(), d_output, outputBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

	//Save image
	cv::imwrite("output" + to_string(blurMatrix_size) + "_gpu.jpg", output);

	// Free the device memory
	SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
}

int main(int argc, char *argv[]){

	string inputImage;
  int blurMatrix_size;
	int blurMatrix_size2;

	if(argc < 2){
		inputImage = default_input_image;
    blurMatrix_size = size1;
    blurMatrix_size2 = size2;
  } else if (argc == 2 ){
    inputImage = argv[1];
    blurMatrix_size = size1;
    blurMatrix_size2 = size2;
  } else {
    inputImage = argv[1];
    if (atoi(argv[2]) % 2 == 0) {
      blurMatrix_size = atoi(argv[2]);
      blurMatrix_size2 = size2;
    } else {
      blurMatrix_size = atoi(argv[2]) + 1;
      blurMatrix_size2 = size2;
    }
  }

	// Read input image from the disk
	cv::Mat input = cv::imread(inputImage, CV_LOAD_IMAGE_COLOR);

	if (input.empty())
	{
		cout << "Image Not Found!" << std::endl;
		cin.get();
		return -1;
	}

	cv::Mat output(input.rows, input.cols, input.type());


	printf("Test on GPU\n");
	//Call the wrapper function
	blur_GPU(input, output, blurMatrix_size);
	blur_GPU(input, output, blurMatrix_size2);


  //Open windows to display images
	namedWindow("Input", cv::WINDOW_NORMAL);
	namedWindow("Output", cv::WINDOW_NORMAL);

	//Show the input and output
	imshow("Input", input);
	imshow("Output", output);

	//Wait for key press
	cv::waitKey();

	return 0;
}
